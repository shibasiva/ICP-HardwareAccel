#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <list>
#include <fstream>
#include <Eigen/Dense>
#include <stdio.h>
#include <pcl/io/pcd_io.h>
#include <pcl/point_types.h>
#include <pcl/point_cloud.h>
#include <pcl/common/transforms.h>
#include <pcl/octree/octree_search.h>

// #include <pcl/gpu/octree/octree.hpp>
// #include <pcl/gpu/containers/device_array.hpp>

// #include "../..pcl/gpu/octree/octree.hpp"
// #include "../pcl/gpu/octree/include/pcl/gpu/octree/octree.hpp"

// #include "hip/hip_runtime.h"
extern "C"{
    #include "./cuPCL/cuOctree/lib/cudaOctree.h"
}

//https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


// #pragma diag_suppress 20012
// add in cmd to supress eigen warnings
//--diag-suppress 20012
using namespace std;
using namespace Eigen;
using namespace pcl;

void ICP(PointCloud<PointXYZ>::Ptr source, PointCloud<PointXYZ>::Ptr reference, map<int, bool>& edge_points);

void GetInfo(void)
{
    hipDeviceProp_t prop;

    int count = 0;
    hipGetDeviceCount(&count);
    printf("\nGPU has cuda devices: %d\n", count);
    for (int i = 0; i < count; ++i) {
        hipGetDeviceProperties(&prop, i);
        
        printf("----device id: %d info----\n", i);
        printf("  GPU : %s \n", prop.name);
        printf("  Capbility: %d.%d\n", prop.major, prop.minor);
        printf("  Global memory: %luMB\n", prop.totalGlobalMem >> 20);
        printf("  Const memory: %luKB\n", prop.totalConstMem  >> 10);
        printf("  SM in a block: %luKB\n", prop.sharedMemPerBlock >> 10);
        printf("  warp size: %d\n", prop.warpSize);
        printf("  threads in a block: %d\n", prop.maxThreadsPerBlock);
        printf("  block dim: (%d,%d,%d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  grid dim: (%d,%d,%d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    }
    printf("\n");
}

__global__ 
void NearestNeighborSearch(
                           float* source, 
                           float* reference, 
                           float* search_indices,
                           int search_indices_len, 
                           int reference_len, 
                           Matrix3f rotation, 
                           Vector3f translation, 
                           int* matched_indices, 
                           float* matched_distances
                           )
{
    if(search_indices_len <= 0) return;
    
    //grid stride loop
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < search_indices_len; i += blockDim.x * gridDim.x) {
        int best_matched_index = 0;
        int sp_index = search_indices[i];
        float s_x = source[sp_index*4];
        float s_y = source[sp_index*4 + 1];
        float s_z = source[sp_index*4 + 2];

        // if(i == 0){
        //     printf("first source pt\n");
        //     printf("%f\n", s_x);
        //     printf("%f\n", s_y);
        //     printf("%f\n", s_z);
        // }
        Vector3f sp (s_x, s_y, s_z);
        Vector3f tsp = rotation * sp + translation;

        float best_matched_distance = ((reference[0] - tsp(0))*(reference[0] - tsp(0)) + 
                                       (reference[1] - tsp(1))*(reference[1] - tsp(1)) + 
                                       (reference[2] - tsp(2))*(reference[2] - tsp(2)));
        
        for(int reference_point_index = 1; reference_point_index < reference_len; reference_point_index++){
            float r_x = reference[reference_point_index*4];
            float r_y = reference[reference_point_index*4 + 1];
            float r_z = reference[reference_point_index*4 + 2];
            // printf("examining point: %f %f %f\n", r_x, r_y, r_z);
            float new_matched_distance = ((r_x - tsp(0))*(r_x - tsp(0))+
                                          (r_y - tsp(1))*(r_y - tsp(1))+
                                          (r_z - tsp(2))*(r_z - tsp(2)));

            if(new_matched_distance < best_matched_distance){
                // printf("i: %i | old distance: %f | new matching distance: %f\n", reference_point_index, best_matched_distance, new_matched_distance);

                best_matched_distance = new_matched_distance;
                best_matched_index = reference_point_index;
                // printf("new matching index: %i\n", reference_point);
            }
        }
        // printf("i: %i | matched index: %i | matched distance: %f\n", i, best_matched_index, best_matched_distance);
        matched_indices[i] = best_matched_index;
        matched_distances[i] = best_matched_distance;   
    }
}

//map the source onto the reference
void ICP(PointCloud<PointXYZ>::Ptr source, PointCloud<PointXYZ>::Ptr reference, map<int, bool>& edge_points)
{
    int max_iter = 100; // max iterations
    double convergence_criteria = 0.001;
    // float resolution = 128.0; 

    Matrix3f total_rotation = Matrix3f::Identity();
    Vector3f total_translation = Vector3f::Zero();


    // //following from this code: https://github.com/NVIDIA-AI-IOT/cuPCL/blob/main/cuOctree/main.cpp
    int regular_priority = 2;
    int higher_priority = 1;
    hipStream_t stream = NULL;
    // hipStreamCreate(&stream);
    hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, regular_priority);
    hipStream_t priority_stream = NULL;
    hipStreamCreateWithPriority(&priority_stream, hipStreamNonBlocking, higher_priority);
    
    //load data onto GPU
    unsigned int nCount = reference->width * reference->height;
    float *referenceData = (float *)reference->points.data();

    unsigned int nDstCount = source->width * source->height;
    float *sourceData = (float *)source->points.data();

    float *cuda_source = NULL;
    gpuErrchk(hipMallocManaged(&cuda_source, sizeof(float) * 4 * nCount, hipMemAttachHost));
    gpuErrchk(hipStreamAttachMemAsync (stream, cuda_source));
    // gpuErrchk(hipMemcpyAsync(cuda_source, sourceData, sizeof(float) * 4 * nCount, hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(cuda_source, sourceData, sizeof(float) * 4 * nCount, hipMemcpyHostToDevice, stream));
    gpuErrchk(hipStreamSynchronize(stream));
    
    float *cuda_reference = NULL;
    gpuErrchk(hipMallocManaged(&cuda_reference, sizeof(float) * 4 *nDstCount, hipMemAttachHost));
    gpuErrchk(hipStreamAttachMemAsync (stream, cuda_reference));
    // gpuErrchk(hipMemcpyAsync(cuda_reference, referenceData, sizeof(float) * 4 * nDstCount, hipMemcpyHostToDevice, stream));
    gpuErrchk(hipMemcpyAsync(cuda_reference, referenceData, sizeof(float) * 4 * nDstCount, hipMemcpyHostToDevice, stream));
    gpuErrchk(hipStreamSynchronize(stream));

    // float resolution = 0.03f;
    vector<int> edge_matched_indices;
    vector<int> nonedge_matched_indices;

    for(int i = 0; i < nDstCount; i++){
        nonedge_matched_indices.push_back(i);
    }

    int blockSize = 1024;
    int numBlocks = (nCount + blockSize - 1) / blockSize;
    // cout<<"block size: "<< blockSize<< endl;
    // cout<<"numBlocks: " << numBlocks << endl;
    float rms_max = 10;
    float rms_min = 0.1;
    float w = 1;

    for (int iter = 0; iter < max_iter; iter++) // iterations
    { 
        cout<<"iter: "<<iter<<endl;
        MatrixXf source_cloud_matrix(3, source->points.size()); //X
        MatrixXf matched_cloud_matrix(3, source->points.size()); //P

        int num_previous_matched_edges = edge_matched_indices.size();
        int num_previous_matched_nonedges = nonedge_matched_indices.size();

        cout <<"edge vector size: " << num_previous_matched_edges << endl;
        cout <<"nonedge vector size: " << num_previous_matched_nonedges << endl;
        float* previous_matched_edges = NULL;
        float* previous_edge_ptr = (float *)edge_matched_indices.data();
        int *edge_matched_indices_results;
        float *edge_matched_distances_results;

        if(num_previous_matched_edges > 0){
            gpuErrchk(hipMallocManaged(&previous_matched_edges, sizeof(float) * num_previous_matched_edges, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (priority_stream, previous_matched_edges));
            gpuErrchk(hipMemcpyAsync(previous_matched_edges, previous_edge_ptr, sizeof(float) *num_previous_matched_edges, hipMemcpyHostToDevice, priority_stream));
            gpuErrchk(hipStreamSynchronize(priority_stream));
           
            gpuErrchk(hipMallocManaged(&edge_matched_indices_results, sizeof(int) * nCount, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (priority_stream, edge_matched_indices_results));
            gpuErrchk(hipMemsetAsync(edge_matched_indices_results, 0, sizeof(unsigned int), priority_stream));
            gpuErrchk(hipStreamSynchronize(priority_stream));
            
            gpuErrchk(hipMallocManaged(&edge_matched_distances_results, sizeof(float) * nCount, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (priority_stream, edge_matched_distances_results));
            gpuErrchk(hipMemsetAsync(edge_matched_distances_results, 0, sizeof(unsigned int), priority_stream));
            gpuErrchk(hipStreamSynchronize(priority_stream));
            NearestNeighborSearch<<<numBlocks, blockSize, 0, priority_stream>>>(
                                                                          cuda_source, 
                                                                          cuda_reference, 
                                                                          previous_matched_edges, 
                                                                          num_previous_matched_edges,
                                                                          nDstCount, 
                                                                          total_rotation, 
                                                                          total_translation, 
                                                                          edge_matched_indices_results,
                                                                          edge_matched_distances_results
                                                                          
                                                                          );
        }
        float* previous_matched_nonedges = NULL;
        float* previous_non_edges_ptr = (float *)nonedge_matched_indices.data();
        int *matched_indices_results;
        float *matched_distances_results;

        if(num_previous_matched_nonedges > 0){
            
            gpuErrchk(hipMallocManaged(&previous_matched_nonedges, sizeof(float) * num_previous_matched_nonedges, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (stream, previous_matched_nonedges));
            gpuErrchk(hipMemcpyAsync(previous_matched_nonedges, previous_non_edges_ptr, sizeof(float) * num_previous_matched_nonedges, hipMemcpyHostToDevice, stream));
            gpuErrchk(hipStreamSynchronize(stream));

            gpuErrchk(hipMallocManaged(&matched_indices_results, sizeof(int) * nCount, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (stream, matched_indices_results));
            gpuErrchk(hipMemsetAsync(matched_indices_results, 0, sizeof(unsigned int), stream));
            gpuErrchk(hipStreamSynchronize(stream));
           
            gpuErrchk(hipMallocManaged(&matched_distances_results, sizeof(float) * nCount, hipMemAttachHost));
            gpuErrchk(hipStreamAttachMemAsync (stream, matched_distances_results));
            gpuErrchk(hipMemsetAsync(matched_distances_results, 0, sizeof(unsigned int), stream));
            gpuErrchk(hipStreamSynchronize(stream));

            
            NearestNeighborSearch<<<numBlocks, blockSize, 0, stream>>>(
                                                                    cuda_source, 
                                                                    cuda_reference, 
                                                                    previous_matched_nonedges, 
                                                                    num_previous_matched_nonedges,
                                                                    nDstCount, 
                                                                    total_rotation, 
                                                                    total_translation, 
                                                                    matched_indices_results, 
                                                                    matched_distances_results
                                                                    );
        }
        
        gpuErrchk(hipStreamSynchronize(stream));
        gpuErrchk(hipStreamSynchronize(priority_stream));
        int num_edge_matched = 0;
        int num_points = 0;
        vector<int> new_edge_matched;
        vector<int> new_nonedge_matched;
        double rms = 0.0;
        cout<<"nDstCount * w: " << nDstCount * w << endl;
        for(int i = 0; i < num_previous_matched_edges + num_previous_matched_nonedges; i++){
            if(num_edge_matched > nDstCount * w){
                break;
            }
            int matched_index = 0;
            if(i <  num_previous_matched_edges){
                matched_index = edge_matched_indices_results[i];
                rms += edge_matched_distances_results[i];
            }
            else{
                matched_index = matched_indices_results[i - num_previous_matched_edges];
                rms += matched_distances_results[i - num_previous_matched_edges];
            }
            
            int selected_index = 0;
            if(edge_points[matched_index]){
                num_edge_matched += 1;
                new_edge_matched.push_back(edge_matched_indices[i]);
                selected_index = edge_matched_indices[i];
            }
            else{
                // cout<<"pushing back on nonedge"<<endl;
                new_nonedge_matched.push_back(nonedge_matched_indices[i - num_previous_matched_edges]);
                selected_index = nonedge_matched_indices[i - num_previous_matched_edges];
            }
            

            Vector3f source_point (source->points[selected_index].x, source->points[selected_index].y, source->points[selected_index].z);
            source_cloud_matrix.col(i) = total_rotation * source_point + total_translation;

            
            Vector3f matched_point (reference->points[selected_index].x, reference->points[selected_index].y, reference->points[selected_index].z);
            matched_cloud_matrix.col(i) = matched_point;
            num_points++;
        }
        edge_matched_indices = new_edge_matched;
        nonedge_matched_indices = new_nonedge_matched;

        source_cloud_matrix = source_cloud_matrix(seqN(0,3), seqN(0,num_previous_matched_edges + num_previous_matched_nonedges));
        matched_cloud_matrix = matched_cloud_matrix(seqN(0,3), seqN(0,num_previous_matched_edges + num_previous_matched_nonedges));
        // for(int i = 0; i < nCount; i ++) {
        //     rms += matched_distances[i]; 
        // }
        // rms /= nCount;
        rms = sqrt(rms/num_points);
        cout<<"rms: " <<rms<<endl;
        if(rms < convergence_criteria){
            cout<<"final rms: " <<rms<<endl;
            break;
        }
        
        //set w
        if(rms < rms_min){ //if less than min, set w to a small number of points
            w = 0.01;
        }
        else if(rms < rms_max){
            w = rms/rms_max*10;
        }
        else{ //rms too large, no confidence
            w = 1;
        }
        
        //cout<<"size of indices: " << sizeof(matched_indices)/sizeof(matched_indices[0])<<endl;
        // for(int i = 0; i < nCount; i++){
        //     cout <<  *(matched_indices + i) << " ";
        // }
        // cout << endl;

        // for(int i = 0; i < nCount; i++){
        //     cout <<  *(matched_distances + i) << " ";
        // }
        // cout << endl;

        // cin.get();   

        Vector3f source_center_of_mass = source_cloud_matrix.rowwise().mean();
        // cout<<source_center_of_mass<<endl;
        source_cloud_matrix = source_cloud_matrix.colwise() - source_center_of_mass; //TODO: check this math: https://stackoverflow.com/questions/42811084/eigen-subtracting-vector-from-matrix-columns
        
        Vector3f matched_center_of_mass = matched_cloud_matrix.rowwise().mean();
        // cout<<matched_center_of_mass<<endl;
        matched_cloud_matrix = matched_cloud_matrix.colwise() - matched_center_of_mass; //TODO: check this math
        // cout<<"found center of masses"<<endl;

        //compute dxd matrix of covariances W
        Matrix3f covariances = Matrix3f::Zero();
        for(int col = 0; col < source_cloud_matrix.cols(); col++){
            covariances = covariances + (source_cloud_matrix.col(col) * matched_cloud_matrix.col(col).transpose());
        }

        // cout<<covariances<<endl;
        // cout<<covariances.rows()<<endl;
        // cout<<covariances.cols()<<endl;

        //compute singular value decomposition U and V
        JacobiSVD<MatrixXf, ComputeThinU | ComputeThinV> svd(covariances); 
        // svd.compute(covariances);
        // cout<<"found U and V"<<endl;

        //compute rotation and translation
        Matrix3f rotation = svd.matrixU() * (svd.matrixV().transpose());
        Vector3f translation = source_center_of_mass - rotation * matched_center_of_mass;

        // cout<< "rotation: " << rotation << endl;
        // cout<< "translation: " << translation << endl;
        // cout<< "source centroid: " << total_rotation *  source_center_of_mass + total_translation << endl;
        // cout<< "matched centroid: " << matched_center_of_mass << endl;
        total_rotation *= rotation.transpose();
        total_translation -= translation;

        // //create transform
        // Matrix4f transform = Matrix4f::Identity();
        // transform.block<3,3>(0,0) = rotation.transpose();
        // transform.block<3,1>(0,3) = -translation;
        // transformPointCloud (*source, *source, transform);

        // hipDeviceSynchronize();
        // hipFree(search);
        // hipFree(index);
        // hipFree(output);
        // hipFree(distance);
        // hipFree(selectedCount);
        if(num_previous_matched_nonedges > 0){
            hipFree(matched_indices_results);
            hipFree(matched_distances_results);
            hipFree(previous_matched_nonedges);
        }
        
        if(num_previous_matched_edges > 0){
            hipFree(edge_matched_indices_results);
            hipFree(edge_matched_distances_results);
            hipFree(previous_matched_edges);
        }

        
    }
    hipFree(cuda_source);
    hipFree(cuda_reference);
    hipStreamDestroy(stream);
    //write result as pcd
    Matrix4f transform = Matrix4f::Identity();
    transform.block<3,3>(0,0) = total_rotation;
    transform.block<3,1>(0,3) = total_translation;
    transformPointCloud (*source, *source, transform);

    *source += *reference;
    pcl::io::savePCDFileASCII ("result.pcd", *source);

    cout<<"saved ICP-GPU output to result.pcd"<<endl;
}

int main(int argc, char** argv){
    
    if(argc != 4){
        cout<<"Usage: ./icp_cpp [pcd source] [pcd reference] [pcd reference edges.txt]"<<endl;
        return 0;
    }
    else{
        PointCloud<PointXYZ>::Ptr source (new PointCloud<PointXYZ>);
        PointCloud<PointXYZ>::Ptr reference (new PointCloud<PointXYZ>);
        
        if (io::loadPCDFile<PointXYZ> (argv[1], *source) == -1){
            string s = argv[1];
            cout<< "Couldn't read file " + s + "\n" << endl;
            return (-1);
        }

        if (io::loadPCDFile<PointXYZ> (argv[2], *reference) == -1){
            string s = argv[2];
            cout<< "Couldn't read file " + s + "\n" << endl;
            return (-1);
        }
        map<int, bool> edge_points;
        std::ifstream infile(argv[3]);
        int a;
        while (infile >> a)
        {
            edge_points[a] = true;
        }
        // cout<<edge_points[4]<<endl;
        // cout<<edge_points[7]<<endl;
        // cout<<edge_points[1]<<endl;
        GetInfo();
        ICP(source, reference, edge_points); 
        return 0;
    }
}